#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <stdio.h>
#include <ctime>
#include <cmath>
#include <cfloat>
#include <vector>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

using namespace std;

#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)

int numberOfPoints = 0;
int numberOfClusters = 0;

__global__ void setup_kernel(hiprandState *state) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    /* Each thread gets different seed, a different sequence
       number, no offset */
    hiprand_init(7+id, id, 0, &state[id]);
}

__global__ void generate_normal_kernel(hiprandState *state, float *xPoints, float *yPoints) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  /* Copy state to local memory for efficiency */
  hiprandState localState = state[i];
  /* Generate pseudo-random uniforms */
  xPoints[i] = hiprand_normal(&localState);
  yPoints[i] = hiprand_normal(&localState);
  /* Copy state back to global memory */
  state[i] = localState;
}

__device__ __host__
float compute_distance(float x1,float x2,float y1,float y2) {
  return sqrtf((x2-x1)*(x2-x1)+(y2-y1)*(y2-y1));
}

__global__
void mapFunction(int * map_data_cluster_index, float *xPoints, float *xCentroids, float *yPoints, float *yCentroids, int numberOfClusters) {
  int j = threadIdx.x + blockIdx.x * blockDim.x;
  int index = 0;
  float minDistance = FLT_MAX;

  for(int i = 0; i < numberOfClusters; i++) {
    float currentDistance = compute_distance(xPoints[j],xCentroids[i],yPoints[j],yCentroids[i]);
    if(currentDistance<minDistance)
    {
      minDistance = currentDistance;
      index = i;
    }
  }
  map_data_cluster_index[j] = index;
}

__global__
void reduce (int *clusterIndex, float *xPoints, float *yPoints, float *sumX, float *sumY, int *nElemsX, int *nElemsY) {
  int j = threadIdx.x + blockIdx.x * blockDim.x;

  int index = clusterIndex[j];

  atomicAdd(&sumX[index],xPoints[j]);
  atomicAdd(&sumY[index],yPoints[j]);
  atomicAdd(&nElemsX[index],1);
  atomicAdd(&nElemsY[index],1);
}

__global__
void calculateNewCentroids (float *xCentroids, float *yCentroids, float *sumX, float *sumY, int *nElemsX, int * nElemsY) {
  int j = threadIdx.x + blockIdx.x * blockDim.x;

  xCentroids[j] = (float) (sumX[j] / nElemsX[j]);
  yCentroids[j] = (float) (sumY[j] / nElemsY[j]);

  //printf ("Number of points in cluster %d is %d\n",j,nElemsX[j]);
}

int main(int agrc, char **argv) {
  hipSetDevice(1);
  hipFree(0);

  srand(time(NULL));

  clock_t tStart = clock();

  numberOfPoints = atoi(argv[1]);

  if (numberOfPoints % 2 == 0) {
    numberOfPoints++;
  }

  int t = 0;

  if (numberOfPoints % 10 != 0) {
    t = 1024;
  } else {
    t = 1000;
  }

  numberOfClusters = atoi(argv[2]);

  int tc = 0;

  if (numberOfClusters % 10 != 0) {
    tc = 64;
  } else {
    tc = 60;
  }

  int maxNumberOfIterations = atoi(argv[3]);

  int n = numberOfPoints / t;
  int nc = numberOfClusters / tc;

  hiprandState *devStates, *devStates2;
  CUDA_CALL(hipMalloc((void **)&devStates, n * t * sizeof(hiprandState)));
  CUDA_CALL(hipMalloc((void **)&devStates2, nc * tc * sizeof(hiprandState)));

  setup_kernel<<<n, t>>>(devStates);
  setup_kernel<<<nc, tc>>>(devStates2);

  thrust::host_vector<int> clusterIndex(numberOfPoints);

  //initialize all the points to belong in sentinel cluster -1
  for (int i = 0; i < clusterIndex.size(); i++) {
    clusterIndex[i] = -1;
  }

  //creating and populating device vectors
  thrust::device_vector<float> xPoints(numberOfPoints);
  thrust::device_vector<float> yPoints(numberOfPoints);
  thrust::device_vector<float> xCentroids(numberOfClusters);
  thrust::device_vector<float> yCentroids(numberOfClusters);
  thrust::device_vector<int> previousIndex(numberOfPoints);
  thrust::device_vector<int> deviceClusterIndex = clusterIndex;

  int *clusterIndexPointer = thrust::raw_pointer_cast(&deviceClusterIndex[0]);
  float *xCentroidsPointer = thrust::raw_pointer_cast(&xCentroids[0]);
  float *xPointsPointer = thrust::raw_pointer_cast(&xPoints[0]);
  float *yCentroidsPointer = thrust::raw_pointer_cast(&yCentroids[0]);
  float *yPointsPointer = thrust::raw_pointer_cast(&yPoints[0]);

  generate_normal_kernel<<<n, t>>>(devStates, xPointsPointer, yPointsPointer);
  generate_normal_kernel<<<nc, tc>>>(devStates2, xCentroidsPointer, yCentroidsPointer);

  bool done = false;
  int i = 0;

  while(i < maxNumberOfIterations) {
    float *sumX, *sumY;
    int *nElemsX, *nElemsY;

    CUDA_CALL(hipMalloc((void **)&sumX, nc * tc * sizeof(float)));
    CUDA_CALL(hipMemset(sumX, 0, nc * tc *  sizeof(float)));

    CUDA_CALL(hipMalloc((void **)&sumY, nc * tc * sizeof(float)));
    CUDA_CALL(hipMemset(sumY, 0, nc * tc *  sizeof(float)));

    CUDA_CALL(hipMalloc((void **)&nElemsX, nc * tc * sizeof(int)));
    CUDA_CALL(hipMemset(nElemsX, 0, nc * tc *  sizeof(int)));

    CUDA_CALL(hipMalloc((void **)&nElemsY, nc * tc * sizeof(int)));
    CUDA_CALL(hipMemset(nElemsY, 0, nc * tc *  sizeof(int)));

    printf("Calling the map function with iteration number %d\n", i);

    mapFunction<<<n, t>>>(clusterIndexPointer,xPointsPointer,xCentroidsPointer,yPointsPointer,yCentroidsPointer, numberOfClusters);
    // Check if the corresponding cluster for each point changed
    done = thrust::equal(deviceClusterIndex.begin(),deviceClusterIndex.end(),previousIndex.begin());
    if (done) {
      printf("Clusters for each point remained the same! Terminating...\n");
      break;
    } else {
      printf("Some points changed their corresponding cluster! Will do another iteration!\n");
    }
    // Copy this cluster index to another value to compare the next index to it
    thrust::copy(deviceClusterIndex.begin(),deviceClusterIndex.end(),previousIndex.begin());
    reduce<<<n, t>>>(clusterIndexPointer, xPointsPointer, yPointsPointer, sumX, sumY, nElemsX, nElemsY);
    calculateNewCentroids<<<nc,tc>>>(xCentroidsPointer, yCentroidsPointer, sumX, sumY, nElemsX, nElemsY);
    i++;
  }

  /*for(int i = 0; i < xCentroids.size(); i++)
  {
    cout << "The X axis value of the centroid number " << i << " is " << xCentroids[i] << endl;
    cout << "The Y axis value of the centroid number " << i << " is " << yCentroids[i] << endl;
  }*/

  printf("Time taken mapping and reducing: %.5fs\n", (double)(clock() - tStart)/CLOCKS_PER_SEC);
}
