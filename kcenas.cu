#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <vector>
#include <stdio.h>
#include <ctime>
#include <cmath>
#include <cfloat>
#include <string>
#include <vector>
#include <sstream>
#include <algorithm>
#include <iterator>
#include <functional>
#include <numeric>
#include <iomanip>
#include <fstream>
#include <limits>
#include <thrust/reduce.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/binary_search.h>
#include <thrust/fill.h>
#include <thrust/transform.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

using namespace std;
#define PI 3.14159265359
//#define numberOfPoints 16
#define NUMBER_OF_CLUSTERS 2
#define MAX_NUMBER_OF_ITERATIONS 20

#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__); \
    return EXIT_FAILURE;}} while(0)

int numberOfPoints = 0;

__global__ void setup_kernel(hiprandState *state) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    /* Each thread gets different seed, a different sequence
       number, no offset */
    hiprand_init(7+id, id, 0, &state[id]);
}

__global__ void generate_normal_kernel(hiprandState *state, float *data_x, float *data_y) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  /* Copy state to local memory for efficiency */
  hiprandState localState = state[i];
  /* Generate pseudo-random uniforms */
  data_x[i] = (hiprand_normal(&localState));
  data_y[i] = (hiprand_normal(&localState));
  /* Copy state back to global memory */
  state[i] = localState;
}

vector<string> &split(string &s, char delim, vector<string> &elems) {
    stringstream ss(s);
    string item;
    while (getline(ss, item, delim)) {
        elems.push_back(item);
    }
    return elems;
}

vector<string> split(string &s, char delim) {
    vector<string> elems;
    split(s, delim, elems);
    return elems;
}

__device__ __host__
float compute_distance(float x1,float x2,float y1,float y2){
  return sqrtf((x2-x1)*(x2-x1)+(y2-y1)*(y2-y1));
}

__global__
void mapFunction(int * map_data_cluster_index,float *data_x,float *cluster_x,float *data_y,float *cluster_y)
{
  int j=threadIdx.x + blockIdx.x * blockDim.x;
  int index=0;
  float minDistance=FLT_MAX;

  for(int i=0;i<NUMBER_OF_CLUSTERS;i++)
  {
    float currDistance=compute_distance(data_x[j],cluster_x[i],data_y[j],cluster_y[i]);
    if(currDistance<minDistance)
    {
      minDistance=currDistance;
      index=i;
    }
  }
  map_data_cluster_index[j]=index;
}

void reduce(thrust::device_vector<int> &data_cluster_index,thrust::device_vector<float> &data_x,thrust::device_vector<float> &data_y,thrust::device_vector<float> &centroids_x,thrust::device_vector<float> &centroids_y, int numberOfPoints)
{
  thrust::device_vector<int> d_data_cluster_index=data_cluster_index;
	thrust::device_vector<float> centroid_sumx(NUMBER_OF_CLUSTERS);
	thrust::device_vector<float> centroid_sumy(NUMBER_OF_CLUSTERS);
	thrust::device_vector<int> new_data_cluster_index(numberOfPoints);
	thrust::fill(centroid_sumx.begin(),centroid_sumx.end(),0);
	thrust::fill(centroid_sumy.begin(),centroid_sumy.end(),0);
	thrust::plus<float> binary_op;
	thrust::equal_to<int> binary_pred;
	thrust::device_vector<int> data_cluster_index_y=data_cluster_index;
	//sorts data_x and data_y by key (groups the points by cluster, which means that the points belonging to the first cluster appear first in the vector)
	thrust::sort_by_key(d_data_cluster_index.begin(),d_data_cluster_index.end(),data_x.begin());
  thrust::sort_by_key(data_cluster_index_y.begin(),data_cluster_index_y.end(),data_y.begin());
	//sums up data_x
	thrust::reduce_by_key(d_data_cluster_index.begin(),d_data_cluster_index.end(),data_x.begin(),new_data_cluster_index.begin(),centroid_sumx.begin(),binary_pred,binary_op);
	//sums up data_y
	thrust::reduce_by_key(d_data_cluster_index.begin(),d_data_cluster_index.end(),data_y.begin(),new_data_cluster_index.begin(),centroid_sumy.begin(),binary_pred,binary_op);
	thrust::device_vector<unsigned int> cluster_begin(numberOfPoints);
  thrust::device_vector<unsigned int> cluster_end(numberOfPoints);
	thrust::counting_iterator<unsigned int>search_begin(0);
	thrust::lower_bound(d_data_cluster_index.begin(),d_data_cluster_index.end(),search_begin,search_begin+numberOfPoints,cluster_begin.begin());
	thrust::upper_bound(d_data_cluster_index.begin(),d_data_cluster_index.end(),search_begin,search_begin+numberOfPoints,cluster_end.begin());
	thrust::device_vector<int> cluster_count_gpu(NUMBER_OF_CLUSTERS);
	thrust::minus<unsigned int> binary_op2;
	thrust::divides<float> binary_op3;
	thrust::transform(cluster_end.begin(),cluster_end.end(),cluster_begin.begin(),cluster_count_gpu.begin(),binary_op2);
	thrust::transform(centroid_sumx.begin(),centroid_sumx.end(),cluster_count_gpu.begin(),centroid_sumx.begin(),binary_op3);
	thrust::transform(centroid_sumy.begin(),centroid_sumy.end(),cluster_count_gpu.begin(),centroid_sumy.begin(),binary_op3);
	cout << "Number of points in the first cluster is " << cluster_count_gpu[0]<<endl;
	cout << "Number of points in the second cluster is " << cluster_count_gpu[1]<<endl;
	centroids_x=centroid_sumx;
	centroids_y=centroid_sumy;
}

int main() {
  using namespace thrust;
  srand(time(NULL));

  numberOfPoints = rand() % 10000 + 1000;
  hiprandState *devStates;
  CUDA_CALL(hipMalloc((void **)&devStates, numberOfPoints * sizeof(hiprandState)));

  setup_kernel<<<numberOfPoints, 1>>>(devStates);


  printf("Generating random numbers based on a normal distribution\n");

  //CUDA_CALL(hipMemcpy(hostResults, devResults, numberOfPoints * sizeof(int), hipMemcpyDeviceToHost));

  host_vector<float> data_x(numberOfPoints);
  host_vector<float> data_y(numberOfPoints);
  host_vector<int> data_cluster_index(numberOfPoints);
  host_vector<float> centroids_x(NUMBER_OF_CLUSTERS);
  host_vector<float> centroids_y(NUMBER_OF_CLUSTERS);
  host_vector<float> centroids_sumx(NUMBER_OF_CLUSTERS);
  host_vector<float> centroids_sumy(NUMBER_OF_CLUSTERS);

  cout << "Initializing the centroids" << endl;

  //initialize all the points to belong in sentinel cluster -1
  for (int i = 0; i < data_cluster_index.size(); i++) {
    data_cluster_index[i]=-1;
  }

  //initialize number of points in all centroids to 0
  for (int i = 0; i < centroids_sumx.size(); i++) {
    centroids_sumx[i]=0;
    centroids_sumy[i]=0;
  }

  cout << "Initializing the data for the initial centroids" << endl;
  centroids_x[0]=0.1;
  centroids_y[0]=0.3;
  centroids_x[1]=0.5;
  centroids_y[1]=0.5;

  //creating and populating device vectors
  thrust::device_vector<float> d_data_x = data_x;
  thrust::device_vector<float> d_data_y = data_y;

  thrust::device_vector<float> d_centroids_x = centroids_x;
  thrust::device_vector<float> d_centroids_y = centroids_y;

  thrust::device_vector<int> prev_index(numberOfPoints);
  thrust::device_vector<int> d_data_cluster_index = data_cluster_index;

  int * data_cluster_index_ptr=thrust::raw_pointer_cast(&d_data_cluster_index[0]);
  float *map_cluster_x=thrust::raw_pointer_cast(&d_centroids_x[0]);
  float *map_data_x=thrust::raw_pointer_cast(&d_data_x[0]);
  float *map_cluster_y=thrust::raw_pointer_cast(&d_centroids_y[0]);
  float *map_data_y=thrust::raw_pointer_cast(&d_data_y[0]);

  generate_normal_kernel<<<numberOfPoints, 1>>>(devStates, map_data_x, map_data_y);

  bool done = false;
  int i = 0;
  while(i < MAX_NUMBER_OF_ITERATIONS) {

    cout << "Calling the map function with iteration number " << i << endl;

    mapFunction<<<numberOfPoints,1>>>(data_cluster_index_ptr,map_data_x,map_cluster_x,map_data_y,map_cluster_y);
    // Check if the corresponding cluster for each point changed
    done = thrust::equal(d_data_cluster_index.begin(),d_data_cluster_index.end(),prev_index.begin());
    if (done) {
      cout << "Clusters for each point remained the same! Terminating..." << endl;
      break;
    } else {
      cout << "Some points changed their corresponding cluster! Will do another iteration!" << endl;
    }
    // Copy this cluster index to another value to compare the next index to it
    thrust::copy(d_data_cluster_index.begin(),d_data_cluster_index.end(),prev_index.begin());
    reduce(d_data_cluster_index,d_data_x,d_data_y,d_centroids_x,d_centroids_y,numberOfPoints);
    i++;
  }

  for(int i=0;i<centroids_x.size();i++)
  {
    cout << "The X axis value of the centroid number " << i << " is " << d_centroids_x[i] << endl;
    cout << "The Y axis value of the centroid number " << i << " is " << d_centroids_y[i] << endl;
  }

  cout << "\n\n\n";
  int n0 = 0;
  int n1 = 0;

  for(int i=0;i<d_data_cluster_index.size();i++)
  {
    if (d_data_cluster_index[i] == 0) {
      n0++;
    } else {
      n1++;
    }
  }

  cout << "Number os points in cluster 0 = " << n0 << endl;
  cout << "Number os points in cluster 1 = " << n1 << endl;
}
